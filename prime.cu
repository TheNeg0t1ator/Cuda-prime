#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdbool.h>
#include "quicksort.cuh"
#include <hip/hip_runtime.h>
__global__ void quicksort(uint64_t arr[][1000], uint64_t low, uint64_t high);

__global__ void generatePrimes(uint64_t limit, uint64_t* primes, int * count) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 2;
    int stride = blockDim.x * gridDim.x;
    
    while (i <= limit) {
        bool isPrimeResult = true;
        
        if (i <= 1) {
            isPrimeResult = false;
        } else {
            for (int j = 2; j * j <= i; j++) {
                if (i % j == 0) {
                    isPrimeResult = false;
                    break;
                }
            }
        }
        
        

        if (isPrimeResult) {
            int index = atomicAdd(count, 1);
            primes[index] = i;
        }
        
        i += stride;
    }
}

int main() { 
    const uint64_t limit = 100000000;
    int blockSize = 256;
    int gridSize = (limit - 2 + blockSize - 1) / blockSize;
    
    uint64_t * primes;
    int * count;
    hipMallocManaged(&primes, limit * sizeof(uint64_t));
    hipMallocManaged(&count, sizeof(int));
    *count = 0;
    
    generatePrimes<<<gridSize, blockSize>>>(limit, primes, count);
    hipDeviceSynchronize();

    remove("output.txt");
    // Open the output file in append mode
    FILE* outputFile = fopen("output.txt", "a+");
    if (outputFile == NULL) {
        printf("Failed to open the output file.\n");
        return 1;
    }
    uint64_t size = *count;
    
    const int primesPerThread = 1000;
    const int numberOfThreads = limit/primesPerThread;

    uint64_t splitprimes[numberOfThreads][primesPerThread];

    for (int i = 0; i < numberOfThreads; i++) { 
        for (int j = 0; j < primesPerThread; j++) {// 1000 is the number of primes in a thread
            splitprimes[i][j] = primes[i * primesPerThread + j];       
        }
    }

    //todo sort the array in parallel
    quicksort_kernel<<<gridSize, blockSize>>>(splitprimes, 0, size - 1);
    // Print the prime numbers to the output file
    for (int i = 0; i < *count; i++) {
        fprintf(outputFile, "%lu\n", primes[i]);
    }
    
    // Close the output file
    fclose(outputFile);
    
    hipFree(primes);
    hipFree(count);

    return 0;
}
