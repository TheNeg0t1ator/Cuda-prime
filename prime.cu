#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdbool.h>
#include "quicksort.cuh"
#include <hip/hip_runtime.h>
__global__ void generatePrimes(uint64_t limit, uint64_t* primes, uint64_t* count) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 2;
    int stride = blockDim.x * gridDim.x;
    
    while (i <= limit) {
        bool isPrimeResult = true;
        
        if (i <= 1) {
            isPrimeResult = false;
        } else {
            for (int j = 2; j * j <= i; j++) {
                if (i % j == 0) {
                    isPrimeResult = false;
                    break;
                }
            }
        }
        
        if (isPrimeResult) {
            uint64_t index = (*count)++;
            primes[index] = i;
        }
        
        i += stride;
    }
}

int main() { 
    const uint64_t limit = 100000000;
    int blockSize = 256;
    int gridSize = (limit - 2 + blockSize - 1) / blockSize;
    
    uint64_t * primes;
    uint64_t* count;
    hipMallocManaged(&primes, limit * sizeof(uint64_t));
    hipMallocManaged(&count, sizeof(uint64_t));
    *count = 0;
    
    generatePrimes<<<gridSize, blockSize>>>(limit, primes, count);
    hipDeviceSynchronize();

    remove("output.txt");
    // Open the output file in append mode
    FILE* outputFile = fopen("output.txt", "a+");
    if (outputFile == NULL) {
        printf("Failed to open the output file.\n");
        return 1;
    }
    
    quicksort(primes, 0, limit - 1);

    // Print the prime numbers to the output file
    for (int i = 0; i < *count; i++) {
        fprintf(outputFile, "%I64d\n", primes[i]);
    }
    
    // Close the output file
    fclose(outputFile);
    
    hipFree(primes);
    hipFree(count);

    return 0;
}
