#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdbool.h>
#include "quicksort.cuh"
#include <hip/hip_runtime.h>
__global__ void generatePrimes(uint64_t limit, uint64_t* primes, int * count) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 2;
    int stride = blockDim.x * gridDim.x;
    
    while (i <= limit) {
        bool isPrimeResult = true;
        
        if (i <= 1) {
            isPrimeResult = false;
        } else {
            for (int j = 2; j * j <= i; j++) {
                if (i % j == 0) {
                    isPrimeResult = false;
                    break;
                }
            }
        }
        
        

        if (isPrimeResult) {
            int index = atomicAdd(count, 1);
            primes[index] = i;
        }
        
        i += stride;
    }
}

int main() { 
    const uint64_t limit = 100000000;
    int blockSize = 256;
    int gridSize = (limit - 2 + blockSize - 1) / blockSize;
    
    uint64_t * primes;
    int * count;
    hipMallocManaged(&primes, limit * sizeof(uint64_t));
    hipMallocManaged(&count, sizeof(int));
    *count = 0;
    
    generatePrimes<<<gridSize, blockSize>>>(limit, primes, count);
    hipDeviceSynchronize();

    remove("output.txt");
    // Open the output file in append mode
    FILE* outputFile = fopen("output.txt", "a+");
    if (outputFile == NULL) {
        printf("Failed to open the output file.\n");
        return 1;
    }
    uint64_t size = *count;
    quicksort(primes, 0, size - 1);

    // Print the prime numbers to the output file
    for (int i = 0; i < *count; i++) {
        fprintf(outputFile, "%lu\n", primes[i]);
    }
    
    // Close the output file
    fclose(outputFile);
    
    hipFree(primes);
    hipFree(count);

    return 0;
}
